#include "hip/hip_runtime.h"
#include "gpu_utils/CudaArray.h"
#include "gpu_utils/CudaDevPtr.h"
#include <cfloat>
#include <climits>

// typedef int (*DeviceFuncPtr)(int, int);

template <typename dtype>
__global__ void ReductionKernel(int num_of_ele, int data_st, int data_gap,
                                devPtr<const dtype> data_arr,
                                devPtr<dtype> output, bool is_max)
{
    CUDA_function;
    // using func = is_max ? std::max : std::min;
    extern __shared__ dtype shared_mem[];

    int tid_global = threadIdx.x + blockIdx.x * blockDim.x;

    int tid_local = threadIdx.x;

    shared_mem[tid_local] = is_max ? (-FLT_MAX) : FLT_MAX;
    // 0. judge illegal
    if (tid_global >= num_of_ele)
        return;
    // 1. load the outer data into shared mem
    int block_id = blockIdx.x;
    // int num_of_thread_per_block = blockDim.x;

    shared_mem[tid_local] = data_arr[data_st + data_gap * tid_global];
    __syncthreads();

    // 2. begin to do reduction

    // do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid_local < s)
        {
            shared_mem[tid_local] =
                is_max ? max(shared_mem[tid_local], shared_mem[tid_local + s])
                       : min(shared_mem[tid_local], shared_mem[tid_local + s]);
            // printf("s = %d, shared_mem[%d] = %.4f\n", s, tid_local,
            //        shared_mem[tid_local]);
        }
        __syncthreads();
    }

    // handle the last warp separately, since for it we can assume that all
    // threads are active
    if (tid_local < 32)
    {

        // when tid_local + 32 exceed the initialzied shared memory?
        volatile float *smem = shared_mem;
        // printf("0 smem[%d] = %.3f\n", tid_local, smem[tid_local]);
        smem[tid_local] = is_max ? (max(smem[tid_local], smem[tid_local + 32]))
                                 : min(smem[tid_local], smem[tid_local + 32]);
        // printf("1 smem[%d] = %.3f\n", tid_local, smem[tid_local]);
        smem[tid_local] = is_max ? (max(smem[tid_local], smem[tid_local + 16]))
                                 : min(smem[tid_local], smem[tid_local + 16]);
        // printf("2 smem[%d] = %.3f\n", tid_local, smem[tid_local]);
        smem[tid_local] = is_max ? (max(smem[tid_local], smem[tid_local + 8]))
                                 : min(smem[tid_local], smem[tid_local + 8]);
        // printf("3 smem[%d] = %.3f\n", tid_local, smem[tid_local]);
        smem[tid_local] = is_max ? (max(smem[tid_local], smem[tid_local + 4]))
                                 : min(smem[tid_local], smem[tid_local + 4]);
        smem[tid_local] = is_max ? (max(smem[tid_local], smem[tid_local + 2]))
                                 : min(smem[tid_local], smem[tid_local + 2]);
        smem[tid_local] = is_max ? (max(smem[tid_local], smem[tid_local + 1]))
                                 : min(smem[tid_local], smem[tid_local + 1]);
        // printf("final smem[%d] = %.3f\n", tid_local, smem[tid_local]);
    }

    // 3. write down
    if (tid_local == 0)
    {
        output[block_id] = shared_mem[0];
    }
}

template <typename dtype>
dtype MinmaxReductionGPU(const cCudaArray<dtype> &data_arr, int ele_st,
                         int ele_gap, int shared_mem_size_bytes, int max_thread,
                         cCudaArray<dtype> &comp_buf, bool is_max)
{
    int ele_bytes = sizeof(dtype);
    // printf("ele_bytes = %d\n", ele_bytes);
    int max_thread_per_block = shared_mem_size_bytes / ele_bytes;
    // printf("max_thread_per_block in hardware = %d\n", max_thread);
    // printf("max_thread_per_block in sm block = %d\n", max_thread_per_block);
    max_thread_per_block = std::min(max_thread, max_thread_per_block);
    // printf("max_thread_per_block = %d\n", max_thread_per_block);

    // 1. determine block_size
    unsigned int thread_per_block = 1;
    while (thread_per_block <= max_thread_per_block)
        thread_per_block <<= 1;
    thread_per_block >>= 1;

    // warp size > 32. warp size = 64.
    // we are rely on at least one single full warp to initialize sm[32] to -inf
    int num_of_ele = (data_arr.Size() - ele_st) / ele_gap +
                     (((data_arr.Size() - ele_st) % ele_gap) != 0);
    while ((thread_per_block >> 1) > num_of_ele &&
           (thread_per_block >> 1) >= 64)
        thread_per_block >>= 1;
    // printf("thread_per_block = %d\n", thread_per_block);

    int num_of_ele_cur = num_of_ele;
    bool is_first_iter = true;

    // printf("num_of_ele_cur = %d\n", num_of_ele_cur);

    int output_size = (num_of_ele_cur / thread_per_block) +
                      (num_of_ele_cur % thread_per_block);
    int total_buf_size = 2 * output_size;
    comp_buf.Resize(total_buf_size);

    int data_st_idx = 0;
    int buf_st_idx = output_size;

    while (true)
    {
        devPtr<const float> data_ptr = nullptr;
        if (is_first_iter)
        {
            data_ptr = data_arr.Ptr();
        }
        else
        {
            data_ptr = comp_buf.Ptr() + data_st_idx;
        };
        devPtr<float> buf_ptr = comp_buf.Ptr() + buf_st_idx;

        output_size = (num_of_ele_cur / thread_per_block) +
                      ((num_of_ele_cur % thread_per_block) != 0);
        int sm_bytes = thread_per_block * sizeof(dtype);
        // printf("output_size after =  %d\n", output_size);
        // printf("sm_bytes = %d\n", sm_bytes);
        ReductionKernel<dtype> CUDA_at_SM(num_of_ele_cur, thread_per_block,
                                          sm_bytes)(
            num_of_ele_cur, (is_first_iter ? ele_st : 0),
            (is_first_iter ? ele_gap : 1), data_ptr, buf_ptr, is_max);
        CUDA_ERR("ReductionKernel");
        num_of_ele_cur = output_size;
        is_first_iter = false;
        // printf("one iter done, num of ele cur = %d\n", num_of_ele_cur);
        if (num_of_ele_cur == 1)
        {
            // std::vector<float> x_cpu;
            // comp_buf.Download(x_cpu, 0);
            // return x_cpu[buf_st_idx];
            std::vector<float> x_cpu_new;
            comp_buf.Download(x_cpu_new, buf_st_idx, buf_st_idx + 1);
            return x_cpu_new[0];
        }

        // swap data and buf
        int tmp = data_st_idx;
        data_st_idx = buf_st_idx;
        buf_st_idx = tmp;
    }
}

template float MinmaxReductionGPU<float>(const cCudaArray<float> &data_arr,
                                         int shared_mem_size_bytes, int ele_st,
                                         int ele_gap, int max_thread,
                                         cCudaArray<float> &comp_buf,
                                         bool is_max);